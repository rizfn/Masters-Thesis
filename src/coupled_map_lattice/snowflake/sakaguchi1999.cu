#include <hip/hip_runtime.h>     // for CUDA-related functions
#include <iostream>   // for std::cout and std::flush
#include <filesystem> // for std::filesystem::path
#include <fstream>    // for std::ofstream
#include <sstream>    // for std::ostringstream

constexpr float Tc = 1;
constexpr float T0 = -0.6;
constexpr int L = 500;
constexpr int N_STEPS = 2000;
constexpr float D = 0.2;
constexpr float C1 = 0.075;
constexpr float C2 = 0.25;
constexpr float g = 0;

// Initialize the data arrays
__global__ void init(float *t1, float *t2, float *x1, float *x2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i * L + j;

    if (i < L && j < L)
    {
        t1[index] = T0;
        x1[index] = 0.0;
        if (i == L / 2 && j == L / 2)
        {
            t1[index] = Tc;
            x1[index] = 1.0;
        }
    }
}

// Diffusion kernel
__global__ void diffuse(float *t1, float *t2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < L - 1 && j > 0 && j < L - 1)
    {
        t2[i * L + j] = t1[i * L + j] + D * (t1[(i + 1) * L + j] + t1[(i - 1) * L + j] + t1[i * L + j + 1] + t1[i * L + j - 1] - 4 * t1[i * L + j]);
    }
}

__global__ void copy(float *src, float *dst)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < L && j < L)
    {
        dst[i * L + j] = src[i * L + j];
    }
}

__global__ void update(float *t1, float *t2, float *x1, float *x2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < L - 1 && j > 0 && j < L - 1)
    {
        // Calculate Noc
        int Noc = (x1[(i + 1) * L + j] >= 1.0) + (x1[i * L + j + 1] >= 1.0) + (x1[(i - 1) * L + j] >= 1.0) + (x1[i * L + j - 1] >= 1.0) +
                  (x1[(i + 1) * L + j + 1] >= 1.0) + (x1[(i + 1) * L + j - 1] >= 1.0) + (x1[(i - 1) * L + j + 1] >= 1.0) + (x1[(i - 1) * L + j - 1] >= 1.0);

        // Calculate Tc(i; j)
        float Tc_ij = Tc + g * (Noc - 3);

        if (t1[i * L + j] < Tc_ij && (x1[(i + 1) * L + j] >= 1.0 || x1[i * L + j + 1] >= 1.0 || x1[(i - 1) * L + j] >= 1.0 || x1[i * L + j - 1] >= 1.0))
        {
            x2[i * L + j] = x1[i * L + j] + C1 * (Tc_ij - t1[i * L + j]);
            t2[i * L + j] = t1[i * L + j] + C2 * (Tc_ij - t1[i * L + j]);
        }
        else
        {
            x2[i * L + j] = x1[i * L + j];
            t2[i * L + j] = t1[i * L + j];
        }
    }
    else if (i == 0 || i == L - 1 || j == 0 || j == L - 1)
    {
        x2[i * L + j] = x1[i * L + j];
        t2[i * L + j] = t1[i * L + j];
    }
}

void run(std::ofstream &file)
{
    // Copy the data to the GPU
    float *d_t1;
    float *d_t2;
    float *d_x1;
    float *d_x2;
    hipMalloc(&d_t1, L * L * sizeof(float));
    hipMalloc(&d_t2, L * L * sizeof(float));
    hipMalloc(&d_x1, L * L * sizeof(float));
    hipMalloc(&d_x2, L * L * sizeof(float));

    // Define the block and grid sizes
    dim3 blockSize(1, 1);
    dim3 gridSize(L, L);

    init<<<gridSize, blockSize>>>(d_t1, d_t2, d_x1, d_x2);
    hipDeviceSynchronize(); // wait for init to finish

    // Run the update function on the GPU
    for (int step = 0; step < N_STEPS; step++)
    {
        diffuse<<<gridSize, blockSize>>>(d_t1, d_t2);
        hipDeviceSynchronize(); // wait for diffuse to finish

        copy<<<gridSize, blockSize>>>(d_t2, d_t1);
        hipDeviceSynchronize(); // wait for copy to finish

        update<<<gridSize, blockSize>>>(d_t1, d_t2, d_x1, d_x2);
        hipDeviceSynchronize(); // wait for update to finish

        copy<<<gridSize, blockSize>>>(d_t2, d_t1);
        copy<<<gridSize, blockSize>>>(d_x2, d_x1);
        hipDeviceSynchronize(); // wait for copy to finish

        std::cout << "Progress: " << std::fixed << std::setprecision(2) << static_cast<double>(step) / (N_STEPS - 1) * 100 << "%\r" << std::flush;
    }

    // Copy the data back to the CPU
    float *t1 = new float[L * L];
    float *x1 = new float[L * L];
    hipMemcpy(t1, d_t1, L * L * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(x1, d_x1, L * L * sizeof(float), hipMemcpyDeviceToHost);

    // Write the x array to the file
    for (int i = 0; i < L; i++)
    {
        for (int j = 0; j < L; j++)
        {
            file << x1[i * L + j];
            if (j < L - 1)
            {
                file << ",";
            }
        }
        file << "\n";
    }

    // Free the GPU memory
    hipFree(d_t1);
    hipFree(d_t2);
    hipFree(d_x1);
    hipFree(d_x2);
}

int main(int argc, char *argv[])
{
    std::string exePath = argv[0];
    std::string exeDir = std::filesystem::path(exePath).parent_path().string();
    std::ostringstream filePathStream;
    filePathStream << exeDir << "/outputs/Tc_" << Tc << "_T0_" << T0 << "_L_" << L << "_D_" << D << "_C1_" << C1 << "_C2_" << C2 << ".csv";
    std::string filePath = filePathStream.str();

    std::ofstream file;
    file.open(filePath);
    run(file);
    file.close();

    return 0;
}