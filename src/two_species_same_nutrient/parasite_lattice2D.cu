// CUDA C++
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <vector>
#include <array>
#include <cstdint>
#include <iostream>
#include <sstream>
#include <fstream>
#include <filesystem>

// Define constants
constexpr double SIGMA = 1;
constexpr double THETA = 0.042;
constexpr double MU = 1;
constexpr double RHO1 = 0.25;
constexpr double RHO2 = 1;
constexpr int L = 1024; // 2^10 = 1024
constexpr int N_STEPS = 10000;

constexpr int BLOCK_LENGTH = 4;

constexpr int EMPTY = 0;
constexpr int NUTRIENT = 1;
constexpr int SOIL = 2;
constexpr int GREEN = 3; // host
constexpr int BLUE = 4;  // parasite


std::vector<int> initLattice(int L)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis_site(0, 4);

    std::vector<int> soil_lattice(L * L);
    for (int i = 0; i < L * L; ++i)
    {
        soil_lattice[i] = dis_site(gen);
    }
    return soil_lattice;
}

__global__ void initCurand(hiprandState *state, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the unique index for the thread
    int index = idx + idy * blockDim.x * gridDim.x;

    // Initialize the RNG state for this thread
    hiprand_init(seed, index, 0, &state[index]);
}

__device__ void getRandomNeighbor(int x, int y, int L, hiprandState *localState, int *nbrX, int *nbrY)
{
    if (hiprand_uniform(localState) < 0.5f)
    {
        *nbrX = (x + (hiprand_uniform(localState) < 0.5f ? -1 : 1) + L) % L;
        *nbrY = y;
    }
    else
    {
        *nbrX = x;
        *nbrY = (y + (hiprand_uniform(localState) < 0.5f ? -1 : 1) + L) % L;
    }
}

__global__ void updateKernel(int *d_lattice, hiprandState *state, int offsetX, int offsetY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the unique index for the thread
    int index = idx + idy * blockDim.x * gridDim.x;

    // Initialize the RNG
    hiprandState localState = state[index]; // Copy the state to local memory for efficiency

    int squareX = (blockIdx.x * BLOCK_LENGTH + offsetX * BLOCK_LENGTH / 2 + threadIdx.x) % L;
    int squareY = (blockIdx.y * BLOCK_LENGTH + offsetY * BLOCK_LENGTH / 2 + threadIdx.y) % L;

    // Select a random site in the 2x2 square (for block length 4)
    int siteX = squareX + hiprand(&localState) % BLOCK_LENGTH / 2;
    int siteY = squareY + hiprand(&localState) % BLOCK_LENGTH / 2;

    // Get the value at the selected site
    int siteValue = d_lattice[siteX * L + siteY];

    // Perform the update
    if (siteValue == EMPTY || siteValue == NUTRIENT)
    {
        // empty or nutrient
        // choose a random neighbour
        int nbrX, nbrY;
        getRandomNeighbor(siteX, siteY, L, &localState, &nbrX, &nbrY);
        int nbrValue = d_lattice[nbrX * L + nbrY];

        if (nbrValue == SOIL)
        {
            // if neighbour is soil
            // fill with soil-filling rate
            if (hiprand_uniform(&localState) < SIGMA)
            {
                d_lattice[siteX * L + siteY] = SOIL;
            }
        }
    }
    else if (siteValue == GREEN)
    {
        // check for death
        if (hiprand_uniform(&localState) < THETA)
        {
            d_lattice[siteX * L + siteY] = EMPTY;
        }
        else
        {
            // move into a neighbour
            int new_siteX, new_siteY;
            getRandomNeighbor(siteX, siteY, L, &localState, &new_siteX, &new_siteY);
            int new_siteValue = d_lattice[new_siteX * L + new_siteY];

            // move the worm
            d_lattice[new_siteX * L + new_siteY] = GREEN;
            d_lattice[siteX * L + siteY] = EMPTY;

            // check if the new site is a nutrient that this worm can consume
            if (new_siteValue == NUTRIENT)
            {
                // reproduce behind you
                if (hiprand_uniform(&localState) < RHO1)
                {
                    d_lattice[siteX * L + siteY] = GREEN;
                }
            }
            // check if the new site is soil
            else if (new_siteValue == SOIL)
            {
                // leave nutrient behind
                if (hiprand_uniform(&localState) < MU)
                {
                    d_lattice[siteX * L + siteY] = NUTRIENT;
                }
            }
            // check if the new site is a worm
            else if (new_siteValue == BLUE || new_siteValue == GREEN)
            {
                // keep both with worms (undo the vacant space in original site)
                d_lattice[siteX * L + siteY] = new_siteValue;
            }
        }
    }
    else if (siteValue == BLUE)
    {
        // check for death
        if (hiprand_uniform(&localState) < THETA)
        {
            d_lattice[siteX * L + siteY] = EMPTY;
        }
        else
        {
            // move into a neighbour
            int new_siteX, new_siteY;
            getRandomNeighbor(siteX, siteY, L, &localState, &new_siteX, &new_siteY);
            int new_siteValue = d_lattice[new_siteX * L + new_siteY];

            // move the worm
            d_lattice[new_siteX * L + new_siteY] = BLUE;
            d_lattice[siteX * L + siteY] = EMPTY;

            // check if the new site is a nutrient that this worm can consume
            if (new_siteValue == NUTRIENT)
            {
                // reproduce behind you
                if (hiprand_uniform(&localState) < RHO2)
                {
                    d_lattice[siteX * L + siteY] = BLUE;
                }
            }
            // check if the new site is a worm
            else if (new_siteValue == BLUE || new_siteValue == GREEN)
            {
                // keep both with worms (undo the vacant space in original site)
                d_lattice[siteX * L + siteY] = new_siteValue;
            }
        }
    }

    // Update the state
    state[index] = localState;
}

void run(std::ofstream &file)
{
    // Initialize the lattice
    std::vector<int> soil_lattice = initLattice(L);

    // Initialize CUDA
    hipSetDevice(0);

    // Allocate memory on the GPU
    int *d_lattice;
    hiprandState *d_state;
    hipMalloc(&d_lattice, L * L * sizeof(int));
    hipMalloc(&d_state, L * L * sizeof(hiprandState));

    // Initialize the RNG states
    initCurand<<<L / BLOCK_LENGTH, L / BLOCK_LENGTH>>>(d_state, time(0));

    // Copy the lattice data to the GPU
    hipMemcpy(d_lattice, soil_lattice.data(), L * L * sizeof(int), hipMemcpyHostToDevice);

    // Define the block and grid sizes
    dim3 blockSize(1, 1);
    dim3 gridSize(L / BLOCK_LENGTH, L / BLOCK_LENGTH);

    // print last cuda error
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(hipError_t) << std::endl;
    }

    // Launch the CUDA kernel for each of the A, B, C, and D squares
    for (int step = 1; step <= N_STEPS; ++step)
    {
        for (int i = 0; i < BLOCK_LENGTH / 2 * BLOCK_LENGTH / 2; ++i) // 1 teration per square in subblock
        {
            updateKernel<<<gridSize, blockSize>>>(d_lattice, d_state, 0, 0); // A squares
            hipDeviceSynchronize();
            updateKernel<<<gridSize, blockSize>>>(d_lattice, d_state, 1, 0); // B squares
            hipDeviceSynchronize();
            updateKernel<<<gridSize, blockSize>>>(d_lattice, d_state, 0, 1); // C squares
            hipDeviceSynchronize();
            updateKernel<<<gridSize, blockSize>>>(d_lattice, d_state, 1, 1); // D squares
            hipDeviceSynchronize();
        }

    // Copy the updated lattice data back to the CPU
    hipMemcpy(soil_lattice.data(), d_lattice, L * L * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < L; ++i)
    {
        for (int j = 0; j < L; ++j)
        {
            int8_t value = static_cast<int8_t>(soil_lattice[i * L + j]);
            file.write(reinterpret_cast<const char *>(&value), sizeof(int8_t));
        }
    }

        std::cout << "Progress: " << std::fixed << std::setprecision(2) << static_cast<double>(step) / N_STEPS * 100 << "%\r" << std::flush;
    }

    // Free the memory allocated on the GPU
    hipFree(d_lattice);
    hipFree(d_state);
}

int main(int argc, char *argv[])
{
    std::string exePath = argv[0];
    std::string exeDir = std::filesystem::path(exePath).parent_path().string();
    std::ostringstream filePathStream;
    // filePathStream << exeDir << "/outputs/confinement/sigma_" << SIGMA << "_theta_" << THETA << "_rhofactor_" << RHO2 / RHO1 << ".tsv";
    // filePathStream << exeDir << "../../docs/data/twospec_samenutrient/lattice_anim_" << SIGMA << "_theta_" << THETA << "_rhofactor_" << RHO2 / RHO1 << ".bin";
    filePathStream << "C:/GitHub/Masters-Thesis/docs/data/twospec_samenutrient/lattice_anim_L_" << L << "_sigma_" << SIGMA << "_theta_" << THETA << "_rhofactor_" << RHO2 / RHO1 << ".bin";
    std::string filePath = filePathStream.str();

    std::ofstream file;
    file.open(filePath);
    run(file);
    file.close();

    return 0;
}