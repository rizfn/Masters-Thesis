// CUDA C++
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <vector>
#include <array>
#include <cmath>
#include <cstdint>
#include <iostream>
#include <sstream>
#include <fstream>
#include <filesystem>

// Define constants
constexpr double P = 0.3;
constexpr int L = 100;
constexpr int N_STEPS = 200;

std::vector<bool> initLattice(int L)
{
    std::vector<bool> lattice(L * L, false); // Initialize with all zeros
    for (int i = 0; i < L * L; i += 20)
    {
        lattice[i] = true; // Set every 20th site to 1
    }
    return lattice;
}

__global__ void initCurand(hiprandState *state, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the unique index for the thread
    int index = idx + idy * blockDim.x * gridDim.x;

    // Initialize the RNG state for this thread
    hiprand_init(seed, index, 0, &state[index]);
}

__global__ void updateKernel(bool *d_lattice, bool *d_latticeUpdated, hiprandState *state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the unique index for the thread
    int index = idx + idy * blockDim.x * gridDim.x;

    // Initialize the RNG
    hiprandState localState = state[index]; // Copy the state to local memory for efficiency

    int nPercolationTrials = 0;

    if (d_lattice[idx + idy * L])
    {
        nPercolationTrials++;
    }

    for (int i = 0; i < 4; ++i)
    {
        int x = idx + (i % 2) * 2 - 1;
        int y = idy + (i / 2) * 2 - 1;

        // Periodic boundary conditions
        x = (x + L) % L;
        y = (y + L) % L;

        // Check if the site is occupied
        if (d_lattice[x + y * L])
        {
            nPercolationTrials++;
        }
    }

    if (nPercolationTrials > 0)
    {
        if (hiprand_uniform(&localState) < 1 - pow(1 - P, nPercolationTrials))
        {
            d_latticeUpdated[idx + idy * L] = true;
        }
    }

    // Update the state
    state[index] = localState;
}

void run(std::ofstream &file)
{
    // Initialize the lattice
    std::vector<bool> soil_lattice = initLattice(L);

    // Initialize CUDA
    hipSetDevice(0);

    // Allocate memory on the GPU
    bool *d_lattice;
    bool *d_latticeUpdated;
    hiprandState *d_state;
    hipMalloc(&d_lattice, L * L * sizeof(bool));
    hipMalloc(&d_latticeUpdated, L * L * sizeof(bool));
    hipMalloc(&d_state, L * L * sizeof(hiprandState));

    // Initialize the RNG states
    initCurand<<<L, L>>>(d_state, time(0));

    // Copy the lattice data to a temporary std::vector<char> for the cudaMemcpy call
    std::vector<char> temp_lattice(soil_lattice.begin(), soil_lattice.end());

    // Copy the lattice data to the GPU
    hipMemcpy(d_lattice, temp_lattice.data(), L * L * sizeof(char), hipMemcpyHostToDevice);

    // Define the block and grid sizes
    dim3 blockSize(1, 1);
    dim3 gridSize(L, L);

    // Launch the CUDA kernel for each of the A, B, C, and D squares
    for (int step = 0; step < N_STEPS; ++step)
    {
        // reset the updated lattice to all zeros
        hipMemset(d_latticeUpdated, 0, L * L * sizeof(bool));

        updateKernel<<<gridSize, blockSize>>>(d_lattice, d_latticeUpdated, d_state);
        hipDeviceSynchronize();

        hipMemcpy(d_lattice, d_latticeUpdated, L * L * sizeof(bool), hipMemcpyDeviceToDevice);

        std::cout << "Progress: " << std::fixed << std::setprecision(2) << static_cast<double>(step) / (N_STEPS - 1) * 100 << "%\r" << std::flush;
    }

    // Copy the lattice data back to the CPU
    hipMemcpy(temp_lattice.data(), d_lattice, L * L * sizeof(char), hipMemcpyDeviceToHost);

    // Write the lattice to the file
    for (int i = 0; i < L * L; ++i)
    {
        file << static_cast<int>(temp_lattice[i]);
        if ((i + 1) % L == 0)
        {
            file << std::endl;
        }
        else
        {
            file << ",";
        }
    }

    // Free the memory on the GPU
    hipFree(d_lattice);
    hipFree(d_latticeUpdated);
    hipFree(d_state);
}

int main(int argc, char *argv[])
{
    std::string exePath = argv[0];
    std::string exeDir = std::filesystem::path(exePath).parent_path().string();
    std::ostringstream filePathStream;
    filePathStream << exeDir << "/outputs/lattice2D/CUDA_p_" << P << "_L_" << L << ".csv";
    std::string filePath = filePathStream.str();

    std::ofstream file;
    file.open(filePath);
    run(file);
    file.close();

    return 0;
}