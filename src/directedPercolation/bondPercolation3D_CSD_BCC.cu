// CUDA C++
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <vector>
#include <array>
#include <unordered_map>
#include <cmath>
#include <cstdint>
#include <iostream>
#include <sstream>
#include <fstream>
#include <filesystem>

// Define constants
constexpr double P = 0.194;
constexpr int L = 128;
constexpr int N_STEPS = 2000;
constexpr int RECORDING_STEP = N_STEPS / 2;
constexpr int RECORDING_INTERVAL = 5;

std::vector<bool> initLattice(int L)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 1);

    std::vector<bool> soil_lattice(L * L * L, false); // Initialize all cells to false
    for (int i = 0; i < L; ++i)
    {
        for (int j = 0; j < L; ++j)
        {
            for (int k = 0; k < L; ++k)
            {
                // Only initialize "odd" cells
                if ((i + j + k) % 2 == 1)
                {
                    soil_lattice[i * L * L + j * L + k] = dis(gen);
                }
            }
        }
    }
    return soil_lattice;
}

__global__ void initCurand(hiprandState *state, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate the unique index for the thread
    int index = idx + idy * blockDim.x * gridDim.x + idz * blockDim.x * gridDim.x * blockDim.y * gridDim.y;

    // Initialize the RNG state for this thread
    hiprand_init(seed, index, 0, &state[index]);
}

__global__ void updateKernel(bool *d_lattice, bool *d_latticeUpdated, double p, hiprandState *state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate the unique index for the thread
    int index = idx + idy * blockDim.x * gridDim.x + idz * blockDim.x * gridDim.x * blockDim.y * gridDim.y;

    // Initialize the RNG
    hiprandState localState = state[index]; // Copy the state to local memory for efficiency

    int nPercolationTrials = 0;

    for (int i = 0; i < 6; ++i)
    {
        int x = idx + (i % 2) * 2 - 1;
        int y = idy + ((i / 2) % 2) * 2 - 1;
        int z = idz + (i / 4) * 2 - 1;

        // Periodic boundary conditions
        x = (x + L) % L;
        y = (y + L) % L;
        z = (z + L) % L;

        // Check if the site is occupied
        if (d_lattice[x + y * L + z * L * L])
        {
            nPercolationTrials++;
        }
    }

    if (nPercolationTrials > 0)
    {
        if (hiprand_uniform(&localState) < 1 - pow(1 - p, nPercolationTrials))
        {
            d_latticeUpdated[idx + idy * L + idz * L * L] = true;
        }
    }

    // Update the state
    state[index] = localState;
}

class UnionFind
{
public:
    UnionFind(int n) : parent(n), rank(n, 0)
    {
        for (int i = 0; i < n; ++i)
            parent[i] = i;
    }

    int find(int i)
    {
        if (parent[i] != i)
            parent[i] = find(parent[i]);
        return parent[i];
    }

    void union_set(int i, int j)
    {
        int ri = find(i), rj = find(j);
        if (ri != rj)
        {
            if (rank[ri] < rank[rj])
                parent[ri] = rj;
            else if (rank[ri] > rank[rj])
                parent[rj] = ri;
            else
            {
                parent[ri] = rj;
                ++rank[rj];
            }
        }
    }

private:
    std::vector<int> parent, rank;
};

std::pair<std::vector<int>, std::vector<int>> get_cluster_sizes(const std::vector<bool> &lattice)
{
    UnionFind uf_filled(L * L * L);
    UnionFind uf_empty(L * L * L);
    for (int i = 0; i < L; ++i)
    {
        for (int j = 0; j < L; ++j)
        {
            for (int k = 0; k < L; ++k)
            {
                int index = (i * L + j) * L + k;
                if (lattice[index])
                {
                    if (lattice[((i - 1 + L) % L * L + j) * L + k])
                        uf_filled.union_set(index, ((i - 1 + L) % L * L + j) * L + k);
                    if (lattice[(i * L + (j - 1 + L) % L) * L + k])
                        uf_filled.union_set(index, (i * L + (j - 1 + L) % L) * L + k);
                    if (lattice[(i * L + j) * L + (k - 1 + L) % L])
                        uf_filled.union_set(index, (i * L + j) * L + (k - 1 + L) % L);
                    if (lattice[((i + 1) % L * L + j) * L + k])
                        uf_filled.union_set(index, ((i + 1) % L * L + j) * L + k);
                    if (lattice[(i * L + (j + 1) % L) * L + k])
                        uf_filled.union_set(index, (i * L + (j + 1) % L) * L + k);
                    if (lattice[(i * L + j) * L + (k + 1) % L])
                        uf_filled.union_set(index, (i * L + j) * L + (k + 1) % L);
                }
                else
                {
                    if (!lattice[((i - 1 + L) % L * L + j) * L + k])
                        uf_empty.union_set(index, ((i - 1 + L) % L * L + j) * L + k);
                    if (!lattice[(i * L + (j - 1 + L) % L) * L + k])
                        uf_empty.union_set(index, (i * L + (j - 1 + L) % L) * L + k);
                    if (!lattice[(i * L + j) * L + (k - 1 + L) % L])
                        uf_empty.union_set(index, (i * L + j) * L + (k - 1 + L) % L);
                    if (!lattice[((i + 1) % L * L + j) * L + k])
                        uf_empty.union_set(index, ((i + 1) % L * L + j) * L + k);
                    if (!lattice[(i * L + (j + 1) % L) * L + k])
                        uf_empty.union_set(index, (i * L + (j + 1) % L) * L + k);
                    if (!lattice[(i * L + j) * L + (k + 1) % L])
                        uf_empty.union_set(index, (i * L + j) * L + (k + 1) % L);
                }
            }
        }
    }

    std::unordered_map<int, int> cluster_sizes_filled;
    std::unordered_map<int, int> cluster_sizes_empty;
    for (int i = 0; i < L; ++i)
    {
        for (int j = 0; j < L; ++j)
        {
            for (int k = 0; k < L; ++k)
            {
                int index = (i * L + j) * L + k;
                if (lattice[index])
                {
                    int root = uf_filled.find(index);
                    ++cluster_sizes_filled[root];
                }
                else
                {
                    int root = uf_empty.find(index);
                    ++cluster_sizes_empty[root];
                }
            }
        }
    }

    std::vector<int> sizes_filled;
    for (const auto &pair : cluster_sizes_filled)
        sizes_filled.push_back(pair.second);

    std::vector<int> sizes_empty;
    for (const auto &pair : cluster_sizes_empty)
        sizes_empty.push_back(pair.second);

    return {sizes_filled, sizes_empty};
}

void run(std::ofstream &file, double p)
{
    std::vector<bool> soil_lattice = initLattice(L * L * L);

    hipSetDevice(0);

    bool *d_lattice;
    bool *d_latticeUpdated;
    hiprandState *d_state;
    hipMalloc(&d_lattice, L * L * L * sizeof(bool));
    hipMalloc(&d_latticeUpdated, L * L * L * sizeof(bool));
    hipMalloc(&d_state, L * L * L * sizeof(hiprandState));

    dim3 blockSize(1, 1, 1);
    dim3 gridSize(L, L, L);

    initCurand<<<gridSize, blockSize>>>(d_state, time(0));

    std::vector<char> temp_lattice(soil_lattice.begin(), soil_lattice.end());
    hipMemcpy(d_lattice, temp_lattice.data(), L * L * L * sizeof(char), hipMemcpyHostToDevice);

    for (int step = 0; step < N_STEPS; ++step)
    {
        hipMemset(d_latticeUpdated, 0, L * L * L * sizeof(bool));

        updateKernel<<<gridSize, blockSize>>>(d_lattice, d_latticeUpdated, p, d_state);
        hipDeviceSynchronize();

        hipMemcpy(d_lattice, d_latticeUpdated, L * L * L * sizeof(bool), hipMemcpyDeviceToDevice);

        if (step >= RECORDING_STEP && step % RECORDING_INTERVAL == 0)
        {

            // Copy lattice data from GPU to CPU
            std::vector<char> lattice_cpu(L * L * L);
            hipMemcpy(lattice_cpu.data(), d_lattice, L * L * L * sizeof(char), hipMemcpyDeviceToHost);
            std::vector<bool> lattice_bool(lattice_cpu.begin(), lattice_cpu.end());

            // Calculate cluster sizes
            auto [sizes_filled, sizes_empty] = get_cluster_sizes(lattice_bool);

            file << step << "\t";
            // Write cluster sizes to a file
            for (size_t i = 0; i < sizes_filled.size(); ++i)
            {
                file << sizes_filled[i];
                if (i != sizes_filled.size() - 1)
                    file << ",";
            }
            file << "\t";
            for (size_t i = 0; i < sizes_empty.size(); ++i)
            {
                file << sizes_empty[i];
                if (i != sizes_empty.size() - 1)
                    file << ",";
            }
            file << "\n";
        }

        std::cout << "Progress: " << std::fixed << std::setprecision(2) << static_cast<double>(step) / (N_STEPS - 1) * 100 << "%\r" << std::flush;
    }

    hipFree(d_lattice);
    hipFree(d_latticeUpdated);
    hipFree(d_state);
}

int main(int argc, char *argv[])
{
    double p = P; // Use the default value P
    if (argc > 1) // If a command-line argument is provided
    {
        p = std::stod(argv[1]); // Convert the first argument to double and use it as p
    }

    std::string exePath = argv[0];
    std::string exeDir = std::filesystem::path(exePath).parent_path().string();
    std::ostringstream filePathStream;
    filePathStream << exeDir << "/outputs/CSD3D/criticalPoints/p_" << p << "_L_" << L << ".tsv";
    std::string filePath = filePathStream.str();

    std::ofstream file;
    file.open(filePath);
    file << "Step\tfilledClusterSizes\temptyClusterSizes\n";
    run(file, p); // Pass p to the run function
    file.close();

    return 0;
}